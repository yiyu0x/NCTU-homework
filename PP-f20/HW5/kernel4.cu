#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float x1, float y1, float x0, float y0, int* output, int width, int height, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height) return;
    // i -> 1600, j -> 1200
    float dx = (x1 - x0) / width;
    float dy = (y1 - y0) / height;

    float x = x0 + i * dx;
    float y = y0 + j * dy;
    int index = (j * width + i);
    output[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
#define N 1600
#define BLOCK_SIZE 64
void hostFE (float x1, float y1, float x0, float y0, int* output, int width, int height, int maxIterations)
{
    int *h_img, *d_img;
    // Locate CPU memory and GPU memory
    h_img = (int*)malloc(width * height * sizeof(int)); 
    hipMalloc((void **)&d_img, width * height * sizeof(int));
    // Copy memory from CPU to GPU
    // cudaMemcpy(d_img, h_img, width * height * sizeof(int), cudaMemcpyHostToDevice);
    // 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(N / BLOCK_SIZE, N / BLOCK_SIZE);
    mandelKernel<<<blockSize, numBlock>>>(x1, y1, x0, y0, d_img, width, height, maxIterations);
    // Sync
    hipDeviceSynchronize();
    // Copy memory from GPU to CPU
    hipMemcpy(h_img, d_img, width * height * sizeof(int), hipMemcpyDeviceToHost);
    // Copy memory from CPU to CPU(answers)
    memcpy(output, h_img, width * height * sizeof(int));
    free(h_img);
    hipFree(d_img);
}
