#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float x1, float y1, float x0, float y0, int* output, int width, int height, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height) return;
    // i -> 1600, j -> 1200
    float dx = (x1 - x0) / width;
    float dy = (y1 - y0) / height;

    float x = x0 + i * dx;
    float y = y0 + j * dy;
    int index = (j * pitch + i);
    output[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
#define N 1600
#define BLOCK_SIZE 64
void hostFE (float x1, float y1, float x0, float y0, int* output, int width, int height, int maxIterations)
{   
    int *h_img = NULL, *d_img = NULL;
    size_t pitch;
    // Locate CPU memory and GPU memory
    hipHostAlloc((void**)&h_img, width * height * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void**)&d_img, &pitch, (size_t)width * sizeof(int), (size_t)height);
    // Copy memory from CPU to GPU
    hipMemcpy2D(d_img, pitch, h_img, width * sizeof(int), width * sizeof(int), height, hipMemcpyHostToDevice);
    // 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(N / BLOCK_SIZE, N / BLOCK_SIZE);
    mandelKernel<<<blockSize, numBlock>>>(x1, y1, x0, y0, d_img, width, height, maxIterations, pitch / sizeof(int));
    // Sync
    hipDeviceSynchronize();
    // // Copy memory from GPU to CPU
    hipMemcpy2D(h_img, width * sizeof(int), d_img, pitch, width * sizeof(int), height, hipMemcpyDeviceToHost);
    // // Copy memory from CPU to CPU(answers)
    memcpy(output, h_img, width * height * sizeof(int));
    hipFree(h_img);
    hipFree(d_img);
}
